#include "hip/hip_runtime.h"
#include "agent_fn/function.cuh"

// not possible to include nvstd::function header, see
// https://github.com/FLAMEGPU/FLAMEGPU2/discussions/1199#discussioncomment-9146551
namespace Action {
enum Action {
    RandomWalk = 0,
    Rest = 1,
    CollectResource = 2,
    MoveToClosestResource = 3,
    EOF = 4,
};
}

FLAMEGPU_DEVICE_FUNCTION float vec2Length(int x, int y) { return sqrtf(((x * x) + (y * y))); }
FLAMEGPU_DEVICE_FUNCTION int findMax(int ar[], int len) {
    int max_index = len;
    int max = 0;
    for (int i = 0; i < len; i++) {
        // printf("[%02d] score[%d]=%d\n", FLAMEGPU->getID(), i, scores[i]);
        if (ar[i] > max) {
            max = ar[i];
            max_index = i;
        }
    }
    return max_index;
}
FLAMEGPU_DEVICE_FUNCTION void printAction(int a) {
    switch (a) {
    case Action::RandomWalk:
        printf("random_walk();\n");
        break;
    case Action::Rest:
        printf("rest();\n");
        break;
    case Action::CollectResource:
        printf("collect_resource();\n");
        break;
    case Action::MoveToClosestResource:
        printf("move_to_closest_resource();\n");
        break;
    case Action::EOF:
    default:
        printf("Action::EOF");
        break;
    }
}

FLAMEGPU_AGENT_FUNCTION(human_behavior, flamegpu::MessageNone, flamegpu::MessageNone) {
    float ap = FLAMEGPU->getVariable<float>("actionpotential");
    int x = FLAMEGPU->getVariable<int>("x");
    int y = FLAMEGPU->getVariable<int>("y");
    int hunger = FLAMEGPU->getVariable<int>("hunger");
    int resources = FLAMEGPU->getVariable<int>("resources");

    auto random_walk = [&]() {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_MOVE");
        int d = 0;
        if (FLAMEGPU->random.uniform<int>(0, 1) == 0) {
            d = 1;
        } else {
            d = -1;
        }
        if (FLAMEGPU->random.uniform<int>(0, 1) == 0) {
            x += d;
        } else {
            y += d;
        }
        int max = FLAMEGPU->environment.getProperty<int>("GRID_SIZE");
        if (x < 0) {
            x = max;
        } else if (y < 0) {
            y = max;
        } else if (x == max) {
            x = 0;
        } else if (y == max) {
            y = 0;
        }
    };
    auto collect_resource = [&]() {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_COLLECT_RESOURCE");
        resources += 1;
    };
    auto move_to_closest_resource = [&]() {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_MOVE");
        float dx = abs((x - FLAMEGPU->getVariable<float>("closest_resource_x")));
        float dy = abs((y - FLAMEGPU->getVariable<float>("closest_resource_y")));
        if (dx > dy) {
            x = (x + ((FLAMEGPU->getVariable<float>("closest_resource_x") - x) / dx));
        } else {
            y = (y + ((FLAMEGPU->getVariable<float>("closest_resource_y") - y) / dy));
        }
    };
    auto rest = [&]() {
        ap += FLAMEGPU->environment.getProperty<float>("AP_PER_TICK_RESTING");
        if (FLAMEGPU->getVariable<int>("is_crowded") == 1) {
            // remove the AP reduction by crowding in case of resting
            ap += FLAMEGPU->environment.getProperty<float>("AP_REDUCTION_BY_CROWDING");
        }
    };
    {
        // changes due to self-perception
        if (FLAMEGPU->getVariable<int>("is_crowded") == 1) {
            ap -= FLAMEGPU->environment.getProperty<float>("AP_REDUCTION_BY_CROWDING");
        }
        hunger += FLAMEGPU->environment.getProperty<int>("HUNGER_PER_TICK");
        if (hunger >= FLAMEGPU->environment.getProperty<int>("HUNGER_STARVED_TO_DEATH")) {
            return flamegpu::DEAD;
        }
        // XXX: strictly speaking food consumption is behavior and should be
        // scored below before executed
        if (resources != 0 &&
            hunger > FLAMEGPU->environment.getProperty<int>("HUNGER_TO_TRIGGER_CONSUMPTION")) {
            resources -= 1;
            hunger -= FLAMEGPU->environment.getProperty<int>("HUNGER_PER_RESOURCE_CONSUMPTION");
        }
    }
    int scores[Action::EOF];
    memset(&scores, 0, Action::EOF * sizeof(int));
    scores[Action::Rest] = 1;
    bool can_collect_resource =
        ap >= FLAMEGPU->environment.getProperty<float>("AP_COLLECT_RESOURCE");
    bool can_move = ap >= FLAMEGPU->environment.getProperty<float>("AP_MOVE");
    if (!(can_move || can_collect_resource)) {
        scores[Action::Rest] = 5;
    }
    if (can_move && FLAMEGPU->getVariable<int>("is_crowded") == 1) {
        scores[Action::RandomWalk] = 10;
    }
    float closest_resource = FLAMEGPU->getVariable<float>("closest_resource");
    if (can_collect_resource &&
        closest_resource <= FLAMEGPU->environment.getProperty<float>("RESOURCE_COLLECTION_RANGE")) {
        scores[Action::CollectResource] = 10;
    }
    if (can_move &&
        closest_resource > FLAMEGPU->environment.getProperty<float>("RESOURCE_COLLECTION_RANGE")) {
        scores[Action::MoveToClosestResource] =
            int(10 - closest_resource * FLAMEGPU->environment.getProperty<float>(
                                            "SCORE_REDUCTION_PER_TILE_DISTANCE"));
    }
    int selected_action = findMax(scores, Action::EOF);
    // printAction(selected_action);
    switch (selected_action) {
    case Action::RandomWalk:
        random_walk();
        break;
    case Action::Rest:
        rest();
        break;
    case Action::CollectResource:
        collect_resource();
        break;
    case Action::MoveToClosestResource:
        move_to_closest_resource();
        break;
    case Action::EOF:
    default:
        printf("[BUG] must not happen\n");
        break;
    }
    FLAMEGPU->setVariable<int>("x", x);
    FLAMEGPU->setVariable<int>("y", y);
    FLAMEGPU->setVariable<float>("actionpotential", ap);
    FLAMEGPU->setVariable<int>("hunger", hunger);
    FLAMEGPU->setVariable<int>("resources", resources);

    return flamegpu::ALIVE;
}
