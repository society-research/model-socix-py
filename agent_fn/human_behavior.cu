#include "hip/hip_runtime.h"
#include "agent_fn/shared.cuh"

// not possible to include nvstd::function header, see
// https://github.com/FLAMEGPU/FLAMEGPU2/discussions/1199#discussioncomment-9146551
namespace Action {
enum Action {
    RandomWalk = 0,
    Rest,
    CollectResource0,
    CollectResource1,
    MoveToClosestResource0,
    MoveToClosestResource1,
    EOF,
};
}

FLAMEGPU_DEVICE_FUNCTION int findMax(int ar[], int len) {
    int max_index = len;
    int max = 0;
    for (int i = 0; i < len; i++) {
        // printf("[%02d] score[%d]=%d\n", FLAMEGPU->getID(), i, scores[i]);
        if (ar[i] > max) {
            max = ar[i];
            max_index = i;
        }
    }
    return max_index;
}
FLAMEGPU_DEVICE_FUNCTION void printAction(int a) {
    switch (a) {
    case Action::RandomWalk:
        printf("random_walk();\n");
        break;
    case Action::Rest:
        printf("rest();\n");
        break;
    case Action::CollectResource0:
        printf("collect_resource(0);\n");
        break;
    case Action::CollectResource1:
        printf("collect_resource(1);\n");
        break;
    case Action::MoveToClosestResource0:
        printf("move_to_closest_resource(0);\n");
        break;
    case Action::MoveToClosestResource1:
        printf("move_to_closest_resource(1);\n");
        break;
    case Action::EOF:
    default:
        printf("Action::EOF");
        break;
    }
}

FLAMEGPU_AGENT_FUNCTION(human_behavior, flamegpu::MessageNone, flamegpu::MessageNone) {
    float ap = FLAMEGPU->getVariable<float>("actionpotential");
    int x = FLAMEGPU->getVariable<int>("x");
    int y = FLAMEGPU->getVariable<int>("y");
    int hunger = FLAMEGPU->getVariable<int>("hunger");
    int resources[N_RESOURCE_TYPES];
    for (int resource_type = 0; resource_type < N_RESOURCE_TYPES; resource_type++) {
        resources[resource_type] =
            FLAMEGPU->getVariable<int, N_RESOURCE_TYPES>("resources", resource_type);
    }

    auto random_walk = [&]() {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_MOVE");
        int d = 0;
        if (FLAMEGPU->random.uniform<int>(0, 1) == 0) {
            d = 1;
        } else {
            d = -1;
        }
        if (FLAMEGPU->random.uniform<int>(0, 1) == 0) {
            x += d;
        } else {
            y += d;
        }
        int max = FLAMEGPU->environment.getProperty<int>("GRID_SIZE");
        if (x < 0) {
            x = max;
        } else if (y < 0) {
            y = max;
        } else if (x == max) {
            x = 0;
        } else if (y == max) {
            y = 0;
        }
    };
    auto collect_resource = [&](int resource_type) {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_COLLECT_RESOURCE");
        resources[resource_type] += 1;
        // store analysis data
        int resource_x =
            FLAMEGPU->getVariable<int, N_RESOURCE_TYPES>("closest_resource_x", resource_type);
        int resource_y =
            FLAMEGPU->getVariable<int, N_RESOURCE_TYPES>("closest_resource_y", resource_type);
        printf("collecting x=%d, y=%d", resource_x, resource_y);
        FLAMEGPU->setVariable<int, 2>("ana_last_resource_location", 0, resource_x);
        FLAMEGPU->setVariable<int, 2>("ana_last_resource_location", 1, resource_y);
    };
    auto move_to_closest_resource = [&](int resource_type) {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_MOVE");
        int closest_x =
            FLAMEGPU->getVariable<int, N_RESOURCE_TYPES>("closest_resource_x", resource_type);
        int closest_y =
            FLAMEGPU->getVariable<int, N_RESOURCE_TYPES>("closest_resource_y", resource_type);
        int dx = abs((x - closest_x));
        int dy = abs((y - closest_y));
        if (dx > dy) {
            x = (x + ((closest_x - x) / dx));
        } else {
            y = (y + ((closest_y - y) / dy));
        }
    };
    auto rest = [&]() {
        ap += FLAMEGPU->environment.getProperty<float>("AP_PER_TICK_RESTING");
        if (FLAMEGPU->getVariable<int>("is_crowded") == 1) {
            // remove the AP reduction by crowding in case of resting
            ap += FLAMEGPU->environment.getProperty<float>("AP_REDUCTION_BY_CROWDING");
        }
    };
    {
        // changes due to self-perception
        if (FLAMEGPU->getVariable<int>("is_crowded") == 1) {
            ap -= FLAMEGPU->environment.getProperty<float>("AP_REDUCTION_BY_CROWDING");
        }
        hunger += FLAMEGPU->environment.getProperty<int>("HUNGER_PER_TICK");
        if (hunger >= FLAMEGPU->environment.getProperty<int>("HUNGER_STARVED_TO_DEATH")) {
            return flamegpu::DEAD;
        }
        // TODO(skep): strictly speaking food consumption is behavior and should be scored below
        // before executed
        // TODO(skep): consume resources[1] as well!
        if (resources[0] != 0 && resources[1] != 0 &&
            hunger > FLAMEGPU->environment.getProperty<int>("HUNGER_TO_TRIGGER_CONSUMPTION")) {
            resources[0] -= 1;
            resources[1] -= 1;
            hunger -= FLAMEGPU->environment.getProperty<int>("HUNGER_PER_RESOURCE_CONSUMPTION");
        }
    }
    int scores[Action::EOF];
    memset(&scores, 0, Action::EOF * sizeof(int));
    scores[Action::Rest] = 1;
    bool can_collect_resource =
        ap >= FLAMEGPU->environment.getProperty<float>("AP_COLLECT_RESOURCE");
    bool can_move = ap >= FLAMEGPU->environment.getProperty<float>("AP_MOVE");
    if (!(can_move || can_collect_resource)) {
        scores[Action::Rest] = 5;
    }
    if (can_move && FLAMEGPU->getVariable<int>("is_crowded") == 1) {
        scores[Action::RandomWalk] = 10;
    }
    for (int resource_type = 0; resource_type < N_RESOURCE_TYPES; resource_type++) {
        float distance_to_resource =
            FLAMEGPU->getVariable<float, N_RESOURCE_TYPES>("closest_resource", resource_type);
        if (can_collect_resource &&
            distance_to_resource <=
                FLAMEGPU->environment.getProperty<float>("RESOURCE_COLLECTION_RANGE")) {
            scores[Action::CollectResource0 + resource_type] =
                10 + (TARGET_RESOURCE_AMOUNT - resources[resource_type]);
        }
        if (can_move &&
            distance_to_resource >
                FLAMEGPU->environment.getProperty<float>("RESOURCE_COLLECTION_RANGE") &&
            distance_to_resource != FLT_MAX) {
            scores[Action::MoveToClosestResource0 + resource_type] =
                int(10 - distance_to_resource * FLAMEGPU->environment.getProperty<float>(
                                                    "SCORE_REDUCTION_PER_TILE_DISTANCE")) +
                /*reduce by resource saturation*/ (TARGET_RESOURCE_AMOUNT -
                                                   resources[resource_type]);
        }
    }
    int selected_action = findMax(scores, Action::EOF);
    // printAction(selected_action);
    switch (selected_action) {
    case Action::RandomWalk:
        random_walk();
        break;
    case Action::Rest:
        rest();
        break;
    case Action::CollectResource0:
        collect_resource(0);
        break;
    case Action::CollectResource1:
        collect_resource(1);
        break;
    case Action::MoveToClosestResource0:
        move_to_closest_resource(0);
        break;
    case Action::MoveToClosestResource1:
        move_to_closest_resource(1);
        break;
    case Action::EOF:
    default:
        printf("[BUG] must not happen\n");
        break;
    }
    FLAMEGPU->setVariable<int>("x", x);
    FLAMEGPU->setVariable<int>("y", y);
    FLAMEGPU->setVariable<float>("actionpotential", ap);
    FLAMEGPU->setVariable<int>("hunger", hunger);
    for (int resource_type = 0; resource_type < N_RESOURCE_TYPES; resource_type++) {
        FLAMEGPU->setVariable<int, N_RESOURCE_TYPES>("resources", resource_type,
                                                     resources[resource_type]);
    }
    return flamegpu::ALIVE;
}
