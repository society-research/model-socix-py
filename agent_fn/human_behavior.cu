#include "hip/hip_runtime.h"
#include "agent_fn/function.cuh"

FLAMEGPU_DEVICE_FUNCTION float vec2Length(int x, int y) { return sqrtf(((x * x) + (y * y))); }

// not possible to include nvstd::function header, see
// https://github.com/FLAMEGPU/FLAMEGPU2/discussions/1199#discussioncomment-9146551
enum class Action : int {
    RandomWalk = 0,
    Rest = 1,
    CollectResource = 2,
    MoveToClosestResource = 3,
    EOF = 4,
};

struct ScoredAction {
    Action action;
    int score;
};

constexpr int NOT_POSSIBLE = 0;

FLAMEGPU_AGENT_FUNCTION(human_behavior, flamegpu::MessageNone, flamegpu::MessageNone) {
    auto ap = FLAMEGPU->getVariable<float>("actionpotential");
    auto x = FLAMEGPU->getVariable<int>("x");
    auto y = FLAMEGPU->getVariable<int>("y");
    auto random_walk = [&]() {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_MOVE");
        auto d = 0;
        if (FLAMEGPU->random.uniform<int>(0, 1) == 0) {
            d = 1;
        } else {
            d = -1;
        }
        if (FLAMEGPU->random.uniform<int>(0, 1) == 0) {
            x += d;
        } else {
            y += d;
        }
        auto max = FLAMEGPU->environment.getProperty<int>("GRID_SIZE");
        if (x < 0) {
            x = max;
        } else if (y < 0) {
            y = max;
        } else if (x == max) {
            x = 0;
        } else if (y == max) {
            y = 0;
        }
    };
    auto collect_resource = [&]() {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_COLLECT_RESOURCE");
        auto resources = FLAMEGPU->getVariable<int>("resources");
        resources += 1;
        FLAMEGPU->setVariable<int>("resources", resources);
    };
    auto move_to_closest_resource = [&]() {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_MOVE");
        auto dx = abs((x - FLAMEGPU->getVariable<float>("closest_resource_x")));
        auto dy = abs((y - FLAMEGPU->getVariable<float>("closest_resource_y")));
        if (dx > dy) {
            x = (x + ((FLAMEGPU->getVariable<float>("closest_resource_x") - x) / dx));
        } else {
            y = (y + ((FLAMEGPU->getVariable<float>("closest_resource_y") - y) / dy));
        }
    };
    auto rest = [&]() {
        ap += FLAMEGPU->environment.getProperty<float>("AP_PER_TICK_RESTING");
        if (FLAMEGPU->getVariable<int>("is_crowded") ==
            1) { // FIXME(goap): only reduce AP by crowding when not resting
            ap += FLAMEGPU->environment.getProperty<float>("AP_REDUCTION_BY_CROWDING");
        }
    };
    {
        if (FLAMEGPU->getVariable<int>("is_crowded") == 1) {
            ap -= FLAMEGPU->environment.getProperty<float>("AP_REDUCTION_BY_CROWDING");
        }
    }
    // XXX: unused right now -- will be used for GOAP-algorithm
    int scores[int(Action::EOF)];
    memset(&scores, 0, int(Action::EOF) * sizeof(int));
    bool can_collect_resource =
        ap >= FLAMEGPU->environment.getProperty<float>("AP_COLLECT_RESOURCE");
    bool can_move = ap >= FLAMEGPU->environment.getProperty<float>("AP_MOVE");
    if (!(can_move || can_collect_resource)) {
        scores[int(Action::Rest)] = 10;
        rest();
    } else if ((can_move && FLAMEGPU->getVariable<int>("is_crowded") == 1)) {
        scores[int(Action::RandomWalk)] = 10;
        random_walk();
    } else if ((can_collect_resource &&
                FLAMEGPU->getVariable<float>("closest_resource") <
                    FLAMEGPU->environment.getProperty<float>("RESOURCE_COLLECTION_RANGE"))) {
        scores[int(Action::CollectResource)] = 10;
        collect_resource();
    } else if ((can_move && FLAMEGPU->getVariable<float>("closest_resource") <=
                                FLAMEGPU->environment.getProperty<float>("HUMAN_MOVE_RANGE"))) {
        scores[int(Action::MoveToClosestResource)] = 10;
        move_to_closest_resource();
    }
    FLAMEGPU->setVariable<int>("x", x);
    FLAMEGPU->setVariable<int>("y", y);
    FLAMEGPU->setVariable<float>("actionpotential", ap);
    return flamegpu::ALIVE;
}
