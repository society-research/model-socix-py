#include "hip/hip_runtime.h"
#include "agent_fn/function.cuh"

FLAMEGPU_DEVICE_FUNCTION float vec2Length(int x, int y) { return sqrtf(((x * x) + (y * y))); }

// not possible to include nvstd::function header, see
// https://github.com/FLAMEGPU/FLAMEGPU2/discussions/1199#discussioncomment-9146551
enum class Action {
    RANDOM_WALK = 1,
};

struct ScoredAction {
    Action action;
    int score;
};

FLAMEGPU_AGENT_FUNCTION(human_behavior, flamegpu::MessageNone, flamegpu::MessageNone) {
    auto ap = FLAMEGPU->getVariable<float>("actionpotential");
    auto x = FLAMEGPU->getVariable<int>("x");
    auto y = FLAMEGPU->getVariable<int>("y");
    auto random_walk = [&]() {
        auto d = 0;
        if (FLAMEGPU->random.uniform<int>(0, 1) == 0) {
            d = 1;
        } else {
            d = -1;
        }
        if (FLAMEGPU->random.uniform<int>(0, 1) == 0) {
            x += d;
        } else {
            y += d;
        }
        auto max = FLAMEGPU->environment.getProperty<int>("GRID_SIZE");
        if (x < 0) {
            x = max;
        } else if (y < 0) {
            y = max;
        } else if (x == max) {
            x = 0;
        } else if (y == max) {
            y = 0;
        }
    };
    auto collect_resource = [&]() {
        auto resources = FLAMEGPU->getVariable<int>("resources");
        resources += 1;
        FLAMEGPU->setVariable<int>("resources", resources);
    };
    // XXX: unused right now -- will be used for GOAP-algorithm
    ScoredAction actions[] = {
        {Action::RANDOM_WALK, 0},
    };
    if (FLAMEGPU->getVariable<int>("is_crowded") == 1) {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_REDUCTION_BY_CROWDING");
    }
    bool can_collect_resource =
        ap >= FLAMEGPU->environment.getProperty<float>("AP_COLLECT_RESOURCE");
    bool can_move = ap >= FLAMEGPU->environment.getProperty<float>("AP_MOVE");
    if (!(can_move || can_collect_resource)) {
        ap += FLAMEGPU->environment.getProperty<float>("AP_PER_TICK_RESTING");
    } else if ((can_move && FLAMEGPU->getVariable<int>("is_crowded") == 1)) {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_MOVE");
        random_walk();
    } else if ((can_collect_resource &&
                FLAMEGPU->getVariable<float>("closest_resource") <
                    FLAMEGPU->environment.getProperty<float>("RESOURCE_COLLECTION_RANGE"))) {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_COLLECT_RESOURCE");
        collect_resource();
    } else if ((can_move && FLAMEGPU->getVariable<float>("closest_resource") <=
                                FLAMEGPU->environment.getProperty<float>("HUMAN_MOVE_RANGE"))) {
        ap -= FLAMEGPU->environment.getProperty<float>("AP_MOVE");
        auto dx = abs((x - FLAMEGPU->getVariable<float>("closest_resource_x")));
        auto dy = abs((y - FLAMEGPU->getVariable<float>("closest_resource_y")));
        if (dx > dy) {
            x = (x + ((FLAMEGPU->getVariable<float>("closest_resource_x") - x) / dx));
        } else {
            y = (y + ((FLAMEGPU->getVariable<float>("closest_resource_y") - y) / dy));
        }
    }
    FLAMEGPU->setVariable<int>("x", x);
    FLAMEGPU->setVariable<int>("y", y);
    FLAMEGPU->setVariable<float>("actionpotential", ap);
    return flamegpu::ALIVE;
}
