#include "hip/hip_runtime.h"
FLAMEGPU_DEVICE_FUNCTION float vec2Length(int x, int y) { return sqrtf(((x * x) + (y * y))); }

FLAMEGPU_AGENT_FUNCTION(human_perception_human_locations, flamegpu::MessageSpatial2D,
                        flamegpu::MessageNone) {
    auto id = FLAMEGPU->getID();
    auto human_x = FLAMEGPU->getVariable<int>("x");
    auto human_y = FLAMEGPU->getVariable<int>("y");
    auto close_humans = 0;
    for (const auto &human : FLAMEGPU->message_in.wrap(human_x, human_y)) {
        if (human.getVariable<int>("id") == id) {
            continue;
        }
        auto other_human_x = human.getVariable<int>("x");
        auto other_human_y = human.getVariable<int>("y");
        if ((human_x == other_human_x && human_y == other_human_y)) {
            close_humans += 1;
        }
    }
    if (close_humans >= FLAMEGPU->environment.getProperty<int>("N_HUMANS_CROWDED")) {
        FLAMEGPU->setVariable<int>("is_crowded", 1);
    } else {
        FLAMEGPU->setVariable<int>("is_crowded", 0);
    }
}
